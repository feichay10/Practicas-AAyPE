/**
 *
 * Universidad de La Laguna
 * Escuela Superior de Ingeniería y Tecnología
 * Grado en Ingeniería Informática
 * Asignatura: Arquitecturas Avanzadas y de Propósito Específico
 * Curso: 4º
 * @file histograma.cu
 * @brief Version 1 histograma en CUDA: Como implementación base (que podremos 
 * mejorar en tiempo o no) se pide crear tantos hilos como elementos de V para 
 * que cada uno se encargue de ir al elemento que le corresponda en V e 
 * incremente la caja correcta en el vector histograma H (posiblemente de forma 
 * atómica).
 * 
 * @version 0.1
 *
 * Compilar y ejecutar con: nvcc histograma.cu -o histograma
 *
 * @date 2023
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>
#include <fstream>

// Function to check the return value of the CUDA runtime API call and exit
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)

#define N 8000000         // Numero de elementos en el vector V
#define M 8                 // Numero de elementos o cajas en el histograma (tamaño del histograma)
#define REPETITIONS 10000      // Numero de repeticiones para el calculo de la media, max y min

#define THREADS_PER_BLOCK 512
#define BLOCKS_PER_GRID ((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

// Variables globales en el device
__device__ int vector_V[N]; // Vector V (vector) de un numero elevado N de elementos enteros aleatorios
__device__ int vector_H[M]; // Vector H (Histograma) que tiene M elementos que representan "cajas"

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
  if (err == hipSuccess) {
    return;
  } 

  std::cerr << statement << " returned " << hipGetErrorString(err) << " (" << err << ") at " << file << ":" << line << std::endl;
  exit (EXIT_FAILURE);
}

/**
 * @brief Function to start the timer
 * 
 * @param start 
 * @param stop 
 */
void startTimer(hipEvent_t *start, hipEvent_t *stop) {
  CUDA_CHECK_RETURN(hipEventCreate(start));
  CUDA_CHECK_RETURN(hipEventCreate(stop));
  CUDA_CHECK_RETURN(hipEventRecord(*start));
}

/**
 * @brief Function to stop the timer and report the elapsed time
 * 
 * @param start 
 * @param stop 
 * @return float 
 */
float stopAndPrintTimer(hipEvent_t *start, hipEvent_t *stop) {
  CUDA_CHECK_RETURN(hipEventRecord(*stop));
  CUDA_CHECK_RETURN(hipEventSynchronize(*stop));

  float milliseconds = 0;
  CUDA_CHECK_RETURN(hipEventElapsedTime(&milliseconds, *start, *stop));

  CUDA_CHECK_RETURN(hipEventDestroy(*start));
  CUDA_CHECK_RETURN(hipEventDestroy(*stop));

  return milliseconds;
}

// ====================================================================================================

/**
 * @brief Kernel para la inicialización del vector V
 * 
 */
__global__ void initVectorV(int random, hiprandState *state) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    hiprand_init(random, i, 0, &state[i]);
    vector_V[i] = hiprand(&state[i]) % 1000;
  }
}

/**
 * @brief Kernel para la inicialización del vector H
 * 
 */
__global__ void initVectorH() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < M) {
    vector_H[i] = 0;
  }
}

/**
 * @brief Kernel para el calculo del histograma con atomicAdd
 * 
 */
__global__ void histogram() {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    atomicAdd(&vector_H[vector_V[i] % M], 1);
  }
}

int main() {
  srand((unsigned)time(NULL)); // Inicializar la semilla para los numeros aleatorios
  int random = rand();         // Generar un numero aleatorio
  hiprandState *devStates;      // Puntero al estado de curand

  int k = 1;

  // Reservar memoria en el device para el estado de curand
  CUDA_CHECK_RETURN(hipMalloc((void **)&devStates, N * sizeof(hiprandState)));

  int host_vector_H[M];           // Vector H en el host
  float elapsedTime[REPETITIONS]; // Array para almacenar los tiempos de ejecución

  std::cout << "Numero de elementos en el vector V: " << N << std::endl;
  std::cout << "Numero de elementos o cajas en el histograma: " << M << std::endl;

  for (int i = 0; i < REPETITIONS; i++) {
    // Inicializar el vector V
    initVectorV<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(random, devStates);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Inicializar el vector H
    initVectorH<<<1, M>>>();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Para calcular el tiempo de ejecución
    hipEvent_t start, stop;
    startTimer(&start, &stop);

    // Calcular el histograma
    histogram<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Detener el temporizador y almacenar el tiempo
    elapsedTime[i] = stopAndPrintTimer(&start, &stop);

    // Copiar el vector H al host
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(host_vector_H, HIP_SYMBOL(vector_H), M * sizeof(int)));

    int acum = 0;
    std::cout << "\nVector H " << k++ << ": \t| ";
    for (int j = 0; j < M; j++) {
      std::cout << host_vector_H[j] << " ";
      acum += host_vector_H[j];
    }
    std::cout << "| \nTotal: " << acum << std::endl;
    std::cout << "Tiempo de ejecución: " << elapsedTime[i] << " ms" << std::endl;
  }

  // Liberar memoria
  CUDA_CHECK_RETURN(hipFree(devStates));

  // Calcular la media, maximo y minimo de los tiempos de ejecución
  float max = 0;
  float min = FLT_MAX;
  float mean = 0;
  for (int i = 0; i < REPETITIONS; i++) {
    mean += elapsedTime[i];
    if (elapsedTime[i] > max) {
      max = elapsedTime[i];
    }
    if (elapsedTime[i] < min) {
      min = elapsedTime[i];
    }
  }

  std::cout << "\n\n==================== RESULTADOS Histograma 1 ====================\n";
  std::cout << "Numero de elementos en el vector V: " << N << std::endl;
  std::cout << "\nSe ha hecho " << REPETITIONS << " repeticiones" << std::endl;
  std::cout << "Tiempo medio:  " << mean / REPETITIONS << " ms" << std::endl;
  std::cout << "Tiempo maximo: " << max << " ms" << std::endl;
  std::cout << "Tiempo minimo: " << min << " ms" << std::endl;

  return 0;
}