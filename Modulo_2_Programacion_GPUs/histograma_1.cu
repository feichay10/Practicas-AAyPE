#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 5000000
#define M 8

// Kernel de operaciones dentro del histograma
__global__ void kernel(int *vector, int *histograma) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int posicion_histograma = 0;
  if (i < N) {
    posicion_histograma = vector[i] % M;  // ValorElementoV mod M
    atomicAdd(&(histograma[posicion_histograma]), 1);
  }
}

// Kernel de inicialización del histograma
__global__ void histograma(int *histo) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    if (i == 0) {
      for (int j = 0; j < M; j++) histo[j] = 0;
    }
  }
}

// Función para mostrar histograma final
void mostrar_histograma(int *hst_vector, int suma) {
  printf("\n\t\t\t\t\tHISTOGRAMA\n\n");
  printf(
      "\n======================================================================"
      "===============================\n");
  for (int i = 1; i <= M; i++) printf("|    %d   | ", i);
  printf("|   TOTAL  | ");
  printf("\n");
  for (int x = 0; x < M; x++) printf("| %d | ", hst_vector[x]);
  printf("| %d  |", suma);

  printf(
      "\n======================================================================"
      "===============================");
}
// rutina principal

int main(void) {
  // vectores de números en host y device
  int *hst_vector = (int *)malloc(N * sizeof(int)); // reservamos memoria en host vector
  int *dev_vector = (int *)malloc(M * sizeof(int)); // reservamos memoria en host histograma
  // vectores de histograma en host y device
  int *hst_histograma;
  int *dev_histograma;
  // errores Cuda
  hipError_t error = hipSuccess;
  // bloques e hilos
  int threadperBlock;
  int blockperGrid;
  // temporizadores
  hipEvent_t start;
  hipEvent_t stop;
  float elapsedTime;
  // comprobación final
  int sum_elements = 0;

  printf("Info: Reservando memoria para los vectores\n");
  // reservamos memoria en host vector
  // hst_vector = (int *)malloc(N * sizeof(int));

  // reservamos memoria en host histograma
  // hst_histograma = (int *)malloc(M * sizeof(int));

  // reservamos memoria en device de vector
    //   error = cudaMalloc((void **)&dev_vector, N * sizeof(int));
    hipMalloc((void **)&dev_vector, N * sizeof(int));

  // reservamos memoria en device de histograma
//   error = cudaMalloc((void **)&dev_histograma, M * sizeof(int));
    hipMalloc((void **)&dev_histograma, M * sizeof(int));

  // inicialización de valores del vector de numeros con valores entre 1 y 8
  srand((int)time(NULL));
  for (int i = 0; i < N; i++) {
    hst_vector[i] = (int)(1 + rand() % (256 - 1));
  }

  // Copiamos los elementos del vector hst_vector en el vector dev_vector
  printf("Info: Copiando elementos de HOST -> DEVICE");
  error = hipMemcpy(dev_vector, hst_vector, N * sizeof(int), hipMemcpyHostToDevice);
 
  error = hipMemcpy(dev_histograma, hst_histograma, M * sizeof(int),hipMemcpyHostToDevice);

  threadperBlock = 512;
  blockperGrid = (N + threadperBlock - 1) / threadperBlock;
  printf("Info: CUDA ejecutara %d hilos y %d bloques\n", threadperBlock, blockperGrid);

  // Ejecutaremos kernel y temporizadores
  printf("Info: Ejecutamos Kernel e iniciamos temporizadores\n");
  hipEventCreate(&start);
  hipEventCreate(&stop);
  // Iniciamos temporizador
  printf("Info: Temporizador iniciado\n");
  hipEventRecord(start, 0);
  histograma<<<blockperGrid, threadperBlock>>>(dev_histograma);
  kernel<<<blockperGrid, threadperBlock>>>(dev_vector, dev_histograma);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
//   error = cudaGetLastError();

  printf("Info: Duración de la creación del histograma completo %.2f sec\n", elapsedTime);
  hipEventDestroy(start);
  hipEventDestroy(stop);

  printf("Info: Copiamos elementos de vectores dev -> host\n");
  error = hipMemcpy(hst_vector, dev_vector, N * sizeof(int),hipMemcpyDeviceToHost);
  error = hipMemcpy(hst_histograma, dev_histograma, M * sizeof(int), hipMemcpyDeviceToHost);

  // comprobamos que la suma de los elementos del histograma corresponde con el
  // numero de elementos
  for (int x = 0; x < M; x++) {
    sum_elements += hst_histograma[x];
  }
  if (sum_elements != N) {  // si no es igual, la ejecucion es incorrecta
    printf("Error, en el histograma hay %d elementos de 5 millones\n",
           sum_elements);
  } else {  // si es igual, la ejecucion es correcta y mostramos el histograma
    printf("\n\nInfo: Ejecución finalizada correctamente :) \n");
    mostrar_histograma(hst_histograma, sum_elements);
  }

  error = hipFree(dev_vector);
  error = hipFree(dev_histograma);
 
  return 0;
}