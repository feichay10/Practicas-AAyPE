/**
 *
 * Universidad de La Laguna
 * Escuela Superior de Ingeniería y Tecnología
 * Grado en Ingeniería Informática
 * Asignatura: Arquitecturas Avanzadas y de Propósito Específico
 * Curso: 4º
 * @file histograma_1.cu
 * @brief Version 2 histograma en CUDA: Realizar un histograma de un vector V de
 * un número elevado N de elementos enteros aleatorios. El histograma consiste 
 * en un vector H que tiene M elementos que representan "cajas". En cada caja se 
 * cuenta el número de veces que ha aparecido un elemento del vector V con el 
 * valor adecuado para asignarlo a esa caja (normalmente cada caja representa un 
 * rango o intervalo de valores). En nuestro caso, para simplificar la asignación
 * del elemento de V a su caja correspondiente del histograma, vamos a realizar 
 * la operación ValorElementoV Módulo M, que nos da directamente el índice de la 
 * caja del histograma a la que pertenecerá ese elemento y cuyo contenido deberemos 
 * incrementar. Se sugiere como N un valor del orden de millones de elementos y 
 * como M, 8 cajas.
 * 
 * Ahora, para la segunda implementación, dividiremos la operación en dos fases. En
 * la primera, en lugar de trabajar sobre un único histograma global, repartiremos 
 * el cálculo realizando un cierto número de histogramas que llamaremos "locales", 
 * cada uno calculado sobre una parte del vector de datos. La idea es reducir el número
 * de hilos que escriben sobre la misma posición del histograma, ya que
 * dicha operación debe ser atómica y se serializan dichos accesos. La segunda fase
 * realizará la suma de los histogramas locales en un único histograma global final.
 * Se debe intentar llevar a cabo esta suma de la forma más paralela o eficiente,
 * posiblemente utilizando el método de reducción.
 * 
 * @version 0.1
 *
 * Compilar y ejecutar con: nvcc histograma.cu -o histograma
 *
 * @date 2023
 *
 */

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <float.h>
#include <iostream>

// Function to check the return value of the CUDA runtime API call and exit
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)

#define N 10000000          // Numero de elementos en el vector V
#define M 8                 // Numero de elementos o cajas en el histograma (tamaño del histograma)
#define REPETITIONS 20      // Numero de repeticiones para el calculo de la media, max y min

#define THREADS_PER_BLOCK 512
#define BLOCKS_PER_GRID ((N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK)

// Variables globales en el device
__device__ int vector_V[N];           // Vector V (vector) de un numero elevado N de elementos enteros aleatorios
__device__ int vector_H[M];           // Vector H (Histograma) que tiene M elementos que representan "cajas"
__shared__ int partial_histogram[M];  // Vector parcial para el histograma

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err) {
  if (err == hipSuccess) {
    return;
  } 

  std::cerr << statement << " returned " << hipGetErrorString(err) << " (" << err << ") at " << file << ":" << line << std::endl;
  exit (EXIT_FAILURE);
}

/**
 * @brief Function to start the timer
 * 
 * @param start 
 * @param stop 
 */
void startTimer(hipEvent_t *start, hipEvent_t *stop) {
  CUDA_CHECK_RETURN(hipEventCreate(start));
  CUDA_CHECK_RETURN(hipEventCreate(stop));
  CUDA_CHECK_RETURN(hipEventRecord(*start));
}

/**
 * @brief Function to stop the timer and report the elapsed time
 * 
 * @param start 
 * @param stop 
 * @return float 
 */
float stopAndPrintTimer(hipEvent_t *start, hipEvent_t *stop) {
  CUDA_CHECK_RETURN(hipEventRecord(*stop));
  CUDA_CHECK_RETURN(hipEventSynchronize(*stop));

  float milliseconds = 0;
  CUDA_CHECK_RETURN(hipEventElapsedTime(&milliseconds, *start, *stop));

  CUDA_CHECK_RETURN(hipEventDestroy(*start));
  CUDA_CHECK_RETURN(hipEventDestroy(*stop));

  return milliseconds;
}

// ====================================================================================================

/**
 * @brief Kernel para la inicialización del vector V
 * 
 */
__global__ void initVectorV(int random, hiprandState *state) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N) {
    hiprand_init(random, i, 0, &state[i]);
    vector_V[i] = hiprand(&state[i]) % 1000;
  }
}

/**
 * @brief Kernel para la inicialización del vector H
 * 
 */
__global__ void initVectorH() {
  
}

/**
 * @brief Kernel para el calculo del histograma por reducción
 * 
 * 1. Trabajar sobre un único histograma global repartiendo el cálculo en histogramas "locales"
 * cada uno calculado sobre una parte del vector de datos. Recudiendo el número de hilos que
 * escriben sobre la misma posición del histograma, ya que dicha operación debe ser atómica y
 * se serializan dichos accesos.
 * 
 * 
 */
__global__ void histogram() {
  // Primera parte:
  // Calcular el histograma local
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) {
    atomicAdd(&partial_histogram[vector_V[i] % M], 1);
  }

}

/**
 * @brief Kernel para la reducción de los histogramas locales en un único histograma global final
 * 
 *  * 2. Realizar la suma de los histogramas locales en un único histograma global final. Se debe
 * intentar llevar a cabo esta suma de la forma más paralela o eficiente, posiblemente utilizando
 * el método de reducción.
 * 
 */
__global__ void reduction() {
  // Segunda parte:
  // Realizar la reducción de los histogramas locales en un único histograma global final
  int i = threadIdx.x;
  int stride = 1; 
  while (stride < M) {
    if (i % (2 * stride) == 0) {
      partial_histogram[i] += partial_histogram[i + stride];
    }
    __syncthreads();
    stride *= 2;
  }

  if (i == 0) {
    atomicAdd(&vector_H[0], partial_histogram[0]);
  }
}

int main() {
  srand((unsigned)time(NULL)); // Inicializar la semilla para los numeros aleatorios
  int random = rand();         // Generar un numero aleatorio
  hiprandState *devStates;      // Puntero al estado de curand

  int k = 1;

  // Reservar memoria en el device para el estado de curand
  CUDA_CHECK_RETURN(hipMalloc((void **)&devStates, N * sizeof(hiprandState)));

  int host_vector_H[M];           // Vector H en el host
  float elapsedTime[REPETITIONS]; // Array para almacenar los tiempos de ejecución

  std::cout << "Numero de elementos en el vector V: " << N << std::endl;
  std::cout << "Numero de elementos o cajas en el histograma: " << M << std::endl;

  for (int i = 0; i < REPETITIONS; i++) {
    // Inicializar el vector V
    initVectorV<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(random, devStates);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Inicializar el vector H
    initVectorH<<<1, M>>>();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Para calcular el tiempo de ejecución
    hipEvent_t start, stop;
    startTimer(&start, &stop);

    // Calcular el histograma
    histogram<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Realizar la reducción de los histogramas locales en un único histograma global final
    reduction<<<1, M>>>();
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    // Detener el temporizador y almacenar el tiempo
    elapsedTime[i] = stopAndPrintTimer(&start, &stop);

    // Copiar el vector H al host
    CUDA_CHECK_RETURN(hipMemcpyFromSymbol(host_vector_H, HIP_SYMBOL(vector_H), M * sizeof(int)));

    int acum = 0;
    std::cout << "\nVector H " << k++ << ": \t| ";
    for (int j = 0; j < M; j++) {
      std::cout << host_vector_H[j] << " ";
      acum += host_vector_H[j];
    }
    std::cout << "| \nTotal: " << acum << std::endl;
    std::cout << "Tiempo de ejecución: " << elapsedTime[i] << " ms" << std::endl;
  }

  // Liberar memoria
  CUDA_CHECK_RETURN(hipFree(devStates));

  // Calcular la media, maximo y minimo de los tiempos de ejecución
  float max = 0;
  float min = FLT_MAX;
  float mean = 0;
  for (int i = 0; i < REPETITIONS; i++) {
    mean += elapsedTime[i];
    if (elapsedTime[i] > max) {
      max = elapsedTime[i];
    }
    if (elapsedTime[i] < min) {
      min = elapsedTime[i];
    }
  }

  std::cout << "\nSe ha hecho " << REPETITIONS << " repeticiones" << std::endl;
  std::cout << "Tiempo medio:  " << mean / REPETITIONS << " ms" << std::endl;
  std::cout << "Tiempo maximo: " << max << " ms" << std::endl;
  std::cout << "Tiempo minimo: " << min << " ms" << std::endl;

  return 0;
}